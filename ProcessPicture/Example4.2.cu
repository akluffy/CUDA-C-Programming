
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

hipError_t addWithCuda(float *Picture, int m, int n);

__global__ void PictureKernel(float *d_Pin, float *d_Pout, int m, int n)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
	int Col = blockIdx.x * blockDim.x + threadIdx.x;

	if((Row < m) && (Col < n)) {
		d_Pout[Row * n + Col] = 2 * d_Pin[Row * n + Col];
	}
}

int main()
{
	// create a picture
	int i, j;
	float picture[9][12];
	for(i = 0; i < 9; i++) {
		for(j = 0; j < 12; j++) {
			picture[i][j] = i * 12 + j;
		}
	}
	// mapping picture data into 1D array
	float *picture_1D;
	picture_1D = (float *)malloc(9*12*sizeof(float));	
	for(i = 0; i < 9; i++) {
		for(j = 0; j < 12; j++) {
			picture_1D[i*12 + j] = picture[i][j];
		}
	}

	// before calling picturekernel
	for(i = 0; i < 9; i++) {
		for(j = 0; j < 12; j++) {
			printf("%1.0f  ", picture_1D[i*12 + j]);
		}
		printf("\n");
	}    

    // call the addwithcuda function.
    hipError_t cudaStatus = addWithCuda(picture_1D, 9, 12);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	printf("\n*******============================================================*********\n");

	// after calling picturekernel
	for(i = 0; i < 9; i++) {
		for(j = 0; j < 12; j++) {
			printf("%1.0f  ", picture_1D[i*12 + j]);
		}
		printf("\n");
	} 

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float *Picture, int m, int n)
{
    float *d_Pin, *d_Pout;
	int Psize = m * n * sizeof(float);
	hipMalloc((void**)&d_Pin, Psize);
	hipMemcpy(d_Pin, Picture, Psize, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_Pout, Psize);
    hipError_t cudaStatus;

	dim3 threadsPerBlocks(16, 16, 1);
	dim3 blocksPerGird(ceil(n/16.0), ceil(m/16.0), 1);
	

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    PictureKernel<<<blocksPerGird, threadsPerBlocks>>>(d_Pin, d_Pout, m, n);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(Picture, d_Pout, Psize, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(d_Pin);
    hipFree(d_Pout);
        
    return cudaStatus;
}
