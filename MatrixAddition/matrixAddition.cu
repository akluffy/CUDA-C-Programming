 #include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

#define M 5
#define N 4

__global__ void MatAdd(float A[M][N], float B[M][N],
                       float C[M][N])
{
    int j = threadIdx.x;
    int i = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}

/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

float A[M][N];
float B[M][N];
float C[M][N];

float (*d_A)[N]; //pointers to arrays of dimension N
float (*d_B)[N];
float (*d_C)[N];

for(int i = 0; i < M; i++) {
    for(int j = 0; j < N; j++) {
        A[i][j] = i;
        B[i][j] = j;
    }
}       

//allocation
hipMalloc((void**)&d_A, (M*N)*sizeof(float));
hipMalloc((void**)&d_B, (M*N)*sizeof(float));
hipMalloc((void**)&d_C, (M*N)*sizeof(float));

//copying from host to device
hipMemcpy(d_A, A, (M*N)*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_B, B, (M*N)*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_C, C, (M*N)*sizeof(float), hipMemcpyHostToDevice);

// Kernel invocation
dim3 threadsPerBlock(N, M);
dim3 numBlocks(N / threadsPerBlock.x, M / threadsPerBlock.y);
MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

//copying from device to host
hipMemcpy(A, (d_A), (M*N)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(B, (d_B), (M*N)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(C, (d_C), (M*N)*sizeof(float), hipMemcpyDeviceToHost);

for(int i = 0; i < M; i++) {
	for(int j = 0; j < N; j++) {
		printf(" %1.f ", A[i][j]);
	}
	printf("\n");
}

printf(" pLUS \n");

for(int i = 0; i < M; i++) {
	for(int j = 0; j < N; j++) {
		printf(" %1.f ", B[i][j]);
	}
	printf("\n");
}

printf("====================\n");

for(int i = 0; i < M; i++) {
	for(int j = 0; j < N; j++) {
		printf(" %1.f ", C[i][j]);
	}
	printf("\n");
}


    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

